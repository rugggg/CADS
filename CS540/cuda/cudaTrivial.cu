// cudaTrivial.cu

#include <hip/hip_runtime.h>
#include <iostream>

__global__ void cudaKernel(int* data) {
    //get thread id
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    //assign to data
    data[i] = i;
}

int main(int argc, char *argv[]){
    //set thread count based on args of blocks and threads
    //ideally would have some named params, but oh well
    int numThreads = 512;
    int numBlocks = 1;
    if(argc < 2){
        std::cout<<"No number of threads or blocks specified"<<std::endl;
    }
    else if(argc == 3){
        std::cout<<"Detected block and thread params..."<<std::endl; 
        numBlocks = std::atoi(argv[1]);
        numThreads = std::atoi(argv[2]);
    }
    else{
        std::cout<<"Detected only 1 arg, assuming it is thread count..."<<std::endl;
        numThreads = std::atoi(argv[1]);
    }
        std::cout<<"Using "<<numBlocks<<" blocks"<<std::endl;
        std::cout<<"Using "<<numThreads<<" threads"<<std::endl;
    int threadCount = numBlocks*numThreads;
    
    int data[threadCount];
    int* d_data;

    //allocate memory on device for int array of numThreads size
    hipMalloc((void **) &d_data, threadCount*sizeof(int));
     
    //invoke kernel
    cudaKernel<<<numBlocks,numThreads>>>(d_data);

    //copy back from device to host
    hipMemcpy(&data,d_data,threadCount*sizeof(int),hipMemcpyDeviceToHost);
    
    //free mem
    hipFree(d_data);
    for(int i = 0; i<threadCount; ++i){
        std::cout<<"Address "<<i<<" :: "<<data[i]<<std::endl;
    }
}




