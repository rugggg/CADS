#include "hip/hip_runtime.h"
//node.cu

#include "node.cuh"
#include <stdlib.h>
#include <vector>
#include <time.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "utils.h"

Node::Node(int x, int y, int numWeights){   
    for (int i = 0; i < numWeights; ++i){
        m_weights.push_back(RandFloat());
    }
    
    m_x = x;
    m_y = y;

}

double Node::calcDistance(const std::vector<double> &compareVector){
    double dis = 0;
    for(int i =0; i < m_weights.size(); ++i){
        dis += (compareVector[i] - m_weights[i]) * (compareVector[i] - m_weights[i]);
    }
    return sqrt(dis);
}


void Node::adjustWeights(const std::vector<double> &target, 
                         const double lambda, 
                         const double influence){

    //std::cout<<getR()<<" "<<getG()<<" "<<getB()<<std::endl;
    
    for(int i=0; i < target.size(); ++i){
      //  std::cout<<"adjust: l:"<<lambda<<"  i:"<<influence<<" ti: "<<target[i]<<" m:"<<m_weights[i]<<std::endl;
        m_weights[i] += lambda * influence * (target[i] - m_weights[i]);
      //  std::cout<<m_weights[i]<<std::endl;
    }
    //std::cout<<"ADJUSTING"<<std::endl;
    //std::cout<<getR()<<" "<<getG()<<" "<<getB()<<std::endl;
    //std::cout<<std::endl;
}

