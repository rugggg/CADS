#include "hip/hip_runtime.h"
//node.cu

#include "node.cuh"
#include <stdlib.h>
#include <vector>
#include <time.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "utils.h"

Node::Node(int x, int y, int numWeights){   
    for (int i = 0; i < numWeights; ++i){
        m_weights.push_back(RandFloat());
    }
    m_x = x;
    m_y = y;
}

double Node::calcDistance(const std::vector<double> &compareVector){
    double dis = 0;
    for(int i =0; i < m_weights.size(); ++i){
        dis += (compareVector[i] - m_weights[i]) * (compareVector[i] - m_weights[i]);
    }
    return sqrt(dis);
}


std::vector<double> Node::adjustWeights(const std::vector<double> &target, 
                         const double lambda, 
                         const double influence){
    for(int i=0; i < target.size(); ++i){
       //m_weights[i] += target[i] - m_weights[i];
       m_weights[i] += lambda * influence * (target[i] - m_weights[i]);
    }
    return m_weights;
}

