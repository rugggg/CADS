#include "hip/hip_runtime.h"
//node.cu

#include "node.cuh"
#include <stdlib.h>
#include <vector>
#include <time.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include "utils.h"

__global__ void cudaAdjustWeights(double *weight, double *target, double lambda, double influence){
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    weight[i] += lambda * influence * (target[i] - weight[i]);
}

Node::Node(int x, int y, int numWeights){   
    for (int i = 0; i < numWeights; ++i){
        m_weights.push_back(RandFloat());
    }
    m_x = x;
    m_y = y;
}

double Node::calcDistance(const std::vector<double> &compareVector){
    double dis = 0;
    for(int i =0; i < m_weights.size(); ++i){
        dis += (compareVector[i] - m_weights[i]) * (compareVector[i] - m_weights[i]);
    }
    return sqrt(dis);
}


__host__ double calcDistanceCuda(thrust::device_vector<double> &compareVector){
    double dis = 0;
    for(int i =0; i < 3; ++i){
        dis += (compareVector[i] - m_weights[i]) * (compareVector[i] - m_weights[i]);
    }
    return sqrt(dis);   
}

__host__ void Node::adjustWeights(const std::vector<double> &target, 
                         const double lambda, 
                         const double influence){
    for(int i=0; i < target.size(); ++i){
       m_weights[i] += lambda * influence * (target[i] - m_weights[i]);
    }
}

__host__ void Node::adjustWeightsCuda(thrust::device_vector<double> target, 
                         const double lambda, 
                         const double influence,
                         const int targetSize){
    
    double *raw_weights = thrust::raw_pointer_cast(&m_weightsCuda[0]);
    double *raw_target = thrust::raw_pointer_cast(&target[0]);
    cudaAdjustWeights<<<1,1>>>(raw_weights, raw_target, lambda, influence);
    thrust::copy(m_weightsCuda.begin(), m_weightsCuda.end(), m_weights.begin());
}

