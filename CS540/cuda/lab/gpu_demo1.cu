
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N (1024*1024)
#define M (1000000)

  __global__ void cudakernel(float *buf)
  {
     /* 
        this line is looking up the address i by taking the thread id (threadIdx.x)
        adding to block id (blockIdx.x), which is multiplied by the block dimensions
        This means that each thread coming into this has a unique ID, then by the line 
        below, the thread is assigned to work on a certain section of the data block
    */
     int i = threadIdx.x + blockIdx.x * blockDim.x; //what is this line doing?  
     buf[i] = 1.0f * i / N;
     for(int j = 0; j < M; j++)
        buf[i] = buf[i] * buf[i] - 0.25f;
  }

  int main()
  {
     float data[N];
     float *d_data; //device pointer

     //allocate memory on GPU
     hipMalloc((void**) &d_data, N*sizeof(float));  
     //invoke kernel with 4096 blocks of 256 threads
     cudakernel<<<4096, 256>>>(d_data); 
     //copy results back to host
     hipMemcpy(data, d_data, N*sizeof(float), hipMemcpyDeviceToHost);
     hipFree(d_data); 

     int input;
     printf("Enter an index: ");
     scanf("%d", &input);
     printf("data[%d] = %f\n", input, data[input]);
  }
