#include "hip/hip_runtime.h"
#include <stdio.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>


#define N (1024*1024)
#define M (1000000)

  __global__ void cudakernel(float *buf)
  {
     /* 
        this line is looking up the address i by taking the thread id (threadIdx.x)
        adding to block id (blockIdx.x), which is multiplied by the block dimensions
        This means that each thread coming into this has a unique ID, then by the line 
        below, the thread is assigned to work on a certain section of the data block
    */
     int i = threadIdx.x + blockIdx.x * blockDim.x; //what is this line doing?  
     buf[i] = 1.0f * i / N;
     for(int j = 0; j < M; j++)
        buf[i] = buf[i] * buf[i] - 0.25f;
  }

  int main()
  {
     thrust::host_vector<float> h_vec(N);
     thrust::device_vector<float> d_vec(N);
     float *raw_d = thrust::raw_pointer_cast(&d_vec[0]);
     //allocate memory on GPU
     //invoke kernel with 4096 blocks of 256 threads
     cudakernel<<<4096, 256>>>(raw_d); 
     //copy results back to host
     thrust::copy(d_vec.begin(), d_vec.end(), h_vec.begin());
        
     int input;
     printf("Enter an index: ");
     scanf("%d", &input);
     printf("data[%d] = %f\n", input, h_vec[input]);
  }
