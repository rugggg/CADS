#include "hip/hip_runtime.h"
#include "som.cuh"
#include "node.cuh"
#include "constants.h"
#include "utils.h"
#include <GL/glut.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

void Som::create(int cellsUp,
                 int cellsAcross,
                 int numIterations)
{
    m_numIterations = numIterations;

    for(int row=0; row<cellsUp; ++row){
        std::vector<Node> m_row;
        for(int col=0; col<cellsAcross; ++col){
            m_row.push_back(Node(row,col,constSizeOfInputVector));
        }
        m_som.push_back(m_row);
    }
    m_mapRadius = max(constWindowWidth, constWindowHeight);
    m_timeConstant = m_numIterations/log(m_mapRadius);
    std::cout<<"tc: "<<m_timeConstant<<"  mr: "<<m_mapRadius<<std::endl;
}

__host__ bool Som::epoch(const std::vector<std::vector<double> > &data){
    if(data[0].size() != constSizeOfInputVector) return false;
    if(m_done) return true;
    if(--m_numIterations > 0){
        int curVector = RandInt(0, data.size()-1);
        m_winningNode = findBestMatch(data[curVector]);
        m_neighborhoodRadius = m_mapRadius * exp(-(double)m_iterationCount/m_timeConstant);

        for(int i=0; i<m_som.size(); ++i){
            for(int n=0; n<m_som[i].size(); ++n){
            double distToNodeSq = (m_winningNode->X()-m_som[i][n].X())*
                                (m_winningNode->X()-m_som[i][n].X())+
                                (m_winningNode->Y()-m_som[i][n].Y())*
                                (m_winningNode->Y()-m_som[i][n].Y());

            double widthSq = m_neighborhoodRadius * m_neighborhoodRadius;
            if(distToNodeSq < (widthSq)){
                m_influence = exp(-(distToNodeSq)/(2*widthSq));
                
                m_som[i][n].adjustWeights(data[curVector], m_lambda, m_influence);
            }
        }
        }
        m_lambda = constStartLearningRate * exp(-(double)m_iterationCount/m_numIterations);
        ++m_iterationCount;
    }
    else{
        m_done = true;
    } 
    return true;
}

__device__ bool Som::cudaEpoch(const thrust::device_vector<thrust::device_vector<double> > &data, int dataSize){
    if(dataSize != constSizeOfInputVector) return false;
    if(m_done) return true;
    if(--m_numIterations > 0){
        int curVector = RandInt(0, data.size()-1);
        //std::cout<<"Current Vector:: "<<std::endl;
        //std::cout<<data[curVector][0]<<" "<<data[curVector][1]<<" "<<data[curVector][2]<<std::endl;
        m_winningNode = findBestMatchCuda(data[curVector]);
        m_neighborhoodRadius = m_mapRadius * exp(-(double)m_iterationCount/m_timeConstant);

        for(int i=0; i<m_som.size(); ++i){
            for(int n=0; n<m_som[i].size(); ++n){
            double distToNodeSq = (m_winningNode->X()-m_som[i][n].X())*
                                (m_winningNode->X()-m_som[i][n].X())+
                                (m_winningNode->Y()-m_som[i][n].Y())*
                                (m_winningNode->Y()-m_som[i][n].Y());

            double widthSq = m_neighborhoodRadius * m_neighborhoodRadius;
            if(distToNodeSq < (widthSq)){
                m_influence = exp(-(distToNodeSq)/(2*widthSq));
                m_som[i][n].adjustWeightsCuda(data[curVector], m_lambda, m_influence, dataSize);
            }
        }
        }
        m_lambda = constStartLearningRate * exp(-(double)m_iterationCount/m_numIterations);
        ++m_iterationCount;
    }
    else{
        m_done = true;
    } 
    return true; 
}

__host__ Node* Som::findBestMatch(const std::vector<double> &vec){
    Node* winner = NULL;
    double lowestDistance = 999999;
    for(int i=0; i<m_som.size(); ++i){
        for(int n=0; n<m_som[i].size(); ++n){
            double dist = m_som[i][n].calcDistance(vec);
            if(dist < lowestDistance){
                lowestDistance = dist;
                winner = &m_som[i][n];
            }
        }
    }
    return winner;
}

__device__ Node* Som::findBestMatchCuda(const thrust::device_vector<double> &vec){
    Node* winner = NULL;
    double lowestDistance = 999999;
    for(int i=0; i<m_som.size(); ++i){
        for(int n=0; n<m_som[i].size(); ++n){
            double dist = m_som[i][n].calcDistanceCuda(vec);
            if(dist < lowestDistance){
                lowestDistance = dist;
                winner = &m_som[i][n];
            }
        }
    }
    return winner;
}


void Som::render(){
//    print();
    
    for (int j=0; j<m_som.size(); ++j){
        for (int i=0; i<m_som[j].size(); ++i){
            glColor3f(m_som[j][i].getR(), m_som[j][i].getG(), m_som[j][i].getB());
            glBegin(GL_QUADS);
            glVertex3f(j, i, 0);            // upper left
            glVertex3f(j, i-1, 0);            // lower left
            glVertex3f(j+1, i-1, 0);            // upper right
            glVertex3f(j+1, i, 0);            // lower right
            glEnd();
        }
    }
}    

void Som::print(){
    std::cout<<std::endl;
    std::cout<<std::endl;
    std::cout<<"==================================================="<<std::endl;
    std::cout<<std::endl;
    for (int j=0; j<m_som.size(); ++j){
        std::vector<Node> row = m_som[j];
        for (int i=0; i<row.size(); ++i){
            std::cout<<"[ "<<row[i].getR()<<", "<<row[i].getG()<<", "<<row[i].getB()<<" ]    ";
        }
        std::cout<<std::endl;
    }
}

void Som::flipDone(){
    m_done = !m_done;
    if(m_done)
        std::cout<<"Stopped Training"<<std::endl;
    else
        std::cout<<"Training..."<<std::endl;
}
