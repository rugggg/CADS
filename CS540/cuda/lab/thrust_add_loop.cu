#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */


#include <thrust/host_vector.h>
#include <thrust/device_vector.h>


#define N   10

__global__ void add( int *a, int *b, int *c ) {
    int tid = blockIdx.x;    // this thread handles the data at its thread id
    if (tid < N)
        c[tid] = a[tid] + b[tid];
}

int main( void ) {
    // allocate the memory on the CPU
    thrust::host_vector<int> h_vec_a(N);
    thrust::host_vector<int> h_vec_b(N);
    thrust::host_vector<int> h_vec_c(N);
    // fill the arrays 'a' and 'b' on the CPU
    for (int i=0; i<N; i++) {
        h_vec_a[i] = -i;
        h_vec_b[i] = i * i;
    }

    // copy the arrays 'a' and 'b' to the GPU
    thrust::device_vector<int> d_vec_a = h_vec_a;
    thrust::device_vector<int> d_vec_b = h_vec_b;
    thrust::device_vector<int> d_vec_c(N);
    int *t_a = thrust::raw_pointer_cast(&d_vec_a[0]);
    int *t_b = thrust::raw_pointer_cast(&d_vec_b[0]);
    int *t_c = thrust::raw_pointer_cast(&d_vec_c[0]);
    //invoke add kernal with correct parameters
    add<<<N,1>>>(t_a, t_b, t_c); //because we use blockIdx.x, we should use 10 blocks, 1 thread each beacuse the operation is not parallelizable, and we need to hit all 10 indexesi

    thrust::copy(d_vec_c.begin(), d_vec_c.end(), h_vec_c.begin());

    // display the results
    for (int i=0; i<N; i++) {
        printf( "%d + %d = %d\n", h_vec_a[i], h_vec_b[i], h_vec_c[i] );
    }

    return 0;
}
