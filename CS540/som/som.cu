#include "hip/hip_runtime.h"
#include "som.cuh"
#include "node.cuh"
#include "constants.h"
#include "utils.h"
#include <GL/glut.h>

void Som::create(int cellsUp,
                 int cellsAcross,
                 int numIterations)
{
    m_numIterations = numIterations;

    for(int row=0; row<cellsUp; ++row){
        std::vector<Node> m_row;
        for(int col=0; col<cellsAcross; ++col){
            m_row.push_back(Node(row,col,constSizeOfInputVector));
        }
        m_som.push_back(m_row);
    }
    m_mapRadius = max(constWindowWidth, constWindowHeight)/2;
    m_timeConstant = m_numIterations/log(m_mapRadius);
}

bool Som::epoch(const std::vector<std::vector<double> > &data){

    if(data[0].size() != constSizeOfInputVector) return false;
    if(m_done) return true;
    if(--m_numIterations > 0){
        int curVector = RandInt(0, data.size()-1);
        //std::cout<<"Current Vector:: "<<std::endl;
        //std::cout<<data[curVector][0]<<" "<<data[curVector][1]<<" "<<data[curVector][2]<<std::endl;
        m_winningNode = findBestMatch(data[curVector]);
        m_neighborhoodRadius = m_mapRadius * exp(-(double)m_iterationCount/m_timeConstant);

        for(int i=0; i<m_som.size(); ++i){
            std::vector<Node> row = m_som[i];
            for(int n=0; n<row.size(); ++n){
            double distToNode = (m_winningNode->X()-row[n].X())*
                                (m_winningNode->X()-row[n].X())+
                                (m_winningNode->Y()-row[n].Y())*
                                (m_winningNode->Y()-row[n].Y());

            double widthSq = m_neighborhoodRadius * m_neighborhoodRadius;
            if(distToNode < (m_neighborhoodRadius * m_neighborhoodRadius)){
                m_influence = exp(-distToNode)/(2*widthSq);
                row[n].adjustWeights(data[curVector], m_lambda, m_influence);
            }
        }
        }
        m_lambda = constStartLearningRate * exp(-(double)m_iterationCount/m_numIterations);
        ++m_iterationCount;
    }
    else{
        m_done = true;
    } 
    return true;
}

Node* Som::findBestMatch(const std::vector<double> &vec){
    Node* winner = NULL;
    double lowestDistance = 999999;
    for(int i=0; i<m_som.size(); ++i){
        std::vector<Node> row = m_som[i];
        for(int n=0; n<row.size(); ++n){
            double dist = row[n].calcDistance(vec);
            if(dist < lowestDistance){
                lowestDistance = dist;
                winner = &row[n];
            }
        }
    }
    return winner;
}

void Som::render(){
//    print();
    for (int j=0; j<m_som.size(); ++j){
        std::vector<Node> row = m_som[j];
        for (int i=0; i<row.size(); ++i){
            std::cout<<"render: "<<j<<","<<i<<std::endl;
            glColor3f(row[i].getR(), row[i].getG(), row[i].getB());
            glBegin(GL_QUADS);
            glVertex3f(j, i, 0);            // upper left
            glVertex3f(j, i-1, 0);            // lower left
            glVertex3f(j+1, i-1, 0);            // upper right
            glVertex3f(j+1, i, 0);            // lower right
            glEnd();
        }
    }
}    

void Som::print(){
    std::cout<<std::endl;
    std::cout<<std::endl;
    std::cout<<"==================================================="<<std::endl;
    std::cout<<std::endl;
    for (int j=0; j<m_som.size(); ++j){
        std::vector<Node> row = m_som[j];
        for (int i=0; i<row.size(); ++i){
            std::cout<<"[ "<<row[i].getR()<<", "<<row[i].getG()<<", "<<row[i].getB()<<" ]    ";
        }
        std::cout<<std::endl;
    }
}

void Som::flipDone(){
    m_done = !m_done;
    if(m_done)
        std::cout<<"Stopped Training"<<std::endl;
    else
        std::cout<<"Training..."<<std::endl;
}
