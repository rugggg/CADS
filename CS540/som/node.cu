#include "hip/hip_runtime.h"
//node.cu

#include "node.cuh"
#include <stdlib.h>
#include <vector>
#include <time.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include "utils.h"

__global__ void cudaAdjustWeights(double *weight, double *target, const double lambda, const double influence){
    int i = (threadIdx.x + blockIdx.x * blockDim.x)-1;
    weight[i] += lambda* influence * (target[i] - weight[i]);
}


Node::Node(int x, int y, int numWeights){   
    for (int i = 0; i < numWeights; ++i){
        double r = RandFloat();
        m_weights.push_back(r);
        m_weightsCuda.push_back(r);
    }
    m_x = x;
    m_y = y;
}

double Node::calcDistance(const std::vector<double> &compareVector){
    double dis = 0;
    for(int i =0; i < m_weights.size(); ++i){
        dis += (compareVector[i] - m_weights[i]) * (compareVector[i] - m_weights[i]);
    }
    return sqrt(dis);
}

__device__ double Node::calcDistanceCuda(double *compareVector, int weightSize, double* cudaWeights){
    double dis = 0;
    for(int i =0; i < weightSize; ++i){
        dis += (compareVector[i] - cudaWeights[i]) * (compareVector[i] - cudaWeights[i]);
    }
    return sqrt(dis);
}


__host__ double Node::adjustWeights(const std::vector<double> &target, 
                         const double lambda, 
                         const double influence){
    double change = 0;
    for(int i=0; i < target.size(); ++i){
       change += abs(lambda * influence * (target[i] - m_weights[i]));
       m_weights[i] += lambda * influence * (target[i] - m_weights[i]);
    }
    return change;
}
__host__ double Node::adjustWeights(const thrust::host_vector<double> &target, 
                         const double lambda, 
                         const double influence){
    double change = 0;
    for(int i=0; i < target.size(); ++i){
       change += abs(lambda * influence * (target[i] - m_weights[i]));
       m_weights[i] += lambda * influence * (target[i] - m_weights[i]);
    }
    return change;
}


__host__ void Node::adjustWeightsCuda(thrust::host_vector<double> target, 
                         const double lambda, 
                         const double influence,
                         const int targetSize){
  
    thrust::device_vector<double> target_d = target;
    thrust::device_vector<double> weights_d = m_weightsCuda; 
    double *raw_weights = thrust::raw_pointer_cast(&weights_d[0]);
    double *raw_target = thrust::raw_pointer_cast(&target_d[0]);
    cudaAdjustWeights<<<3,1>>>(raw_weights, raw_target, lambda, influence);
    thrust::copy(weights_d.begin(), weights_d.end(), m_weightsCuda.begin());
    thrust::copy(weights_d.begin(), weights_d.end(), m_weights.begin());
}

