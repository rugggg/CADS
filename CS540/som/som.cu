#include "hip/hip_runtime.h"
#include "som.cuh"
#include "node.cuh"
#include "constants.h"
#include "utils.h"
#include <GL/glut.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/extrema.h>

__global__ void cudaEpochKernel(){
    
}

//calc distance from a given weight target to all vectors
__global__ void cudaGetDistance(double *targetVec, double *som, double *weights){
       //the som is 3x longer than the weights 
       //targetVec is fixed at length 3
       //let's say we allocate a thread for each weight entry
       int idx = (threadIdx.x + blockIdx.x * blockDim.x)-1;
        
       double dis = 0;
       for(int i =0; i < 3; ++i){
            dis += (targetVec[i] - som[idx+i]) * (targetVec[i] - som[idx+i]);
       }
       weights[idx] = sqrt(dis); 
}


void Som::create(int cellsUp,
                 int cellsAcross,
                 int numIterations)
{
    m_numIterations = numIterations;

    for(int row=0; row<cellsUp; ++row){
        std::vector<Node> m_row;
        for(int col=0; col<cellsAcross; ++col){
            m_row.push_back(Node(row,col,constSizeOfInputVector));
        }
        m_som.push_back(m_row);
    }
    m_mapRadius = max(constWindowWidth, constWindowHeight);
    m_timeConstant = m_numIterations/log(m_mapRadius);
    std::cout<<"tc: "<<m_timeConstant<<"  mr: "<<m_mapRadius<<std::endl;
}

__host__ bool Som::cudaEpoch(std::vector<std::vector <double> > data){
    if(data[0].size() != constSizeOfInputVector) return false;
    if(m_done) return true;
    if(--m_numIterations > 0){
        int curVector = RandInt(0, data.size()-1);
        double *data_raw = thrust::raw_pointer_cast(&data[curVector][0]);
        //we can find the best match by using cuda to calc all distances, and get min
        //flatten the som to be 1d ordered in chunks of 5[r,g,b,x,y]
         thrust::host_vector<double> flat_som;
         for (int r=0; r < m_som.size(); ++r){
            for (int c=0; c <m_som[r].size(); ++c){
              flat_som.push_back(m_som[r][c].getR());
              flat_som.push_back(m_som[r][c].getG());
              flat_som.push_back(m_som[r][c].getB());
              flat_som.push_back(m_som[r][c].X());
              flat_som.push_back(m_som[r][c].Y());
            }
         }
         std::cout<<"gate 1"<<std::endl;
         //allocate to device
         thrust::device_vector<double> som_d = flat_som;
         double *som_raw = thrust::raw_pointer_cast(&som_d[0]);

         std::cout<<"gate 2"<<std::endl;
         thrust::host_vector<double> distances(m_som.size()*m_som.size());
         thrust::device_vector<double> distances_d = distances;
         double *raw_distances = thrust::raw_pointer_cast(&som_d[0]);
         std::cout<<"gate 3"<<std::endl;
         cudaGetDistance<<<100,1>>>(data_raw, som_raw, raw_distances);
         thrust::copy(distances_d.begin(), distances_d.end(), distances.begin());

         std::cout<<"gate 4"<<std::endl;
         thrust::host_vector<double>::iterator iter = thrust::min_element(distances.begin(), distances.end());
         std::cout<<"gate 5"<<std::endl;
         unsigned int position = iter - distances.begin();
         std::cout<<"gate 6"<<std::endl;
         double min_val = *iter;
         m_neighborhoodRadius = m_mapRadius * exp(-(double)m_iterationCount/m_timeConstant);

        //lookup winner in m_som from position
         m_winningNode = &m_som[position/constNumCellsAcross][position%constNumCellsAcross];
         m_neighborhoodRadius = m_mapRadius * exp(-(double)m_iterationCount/m_timeConstant);

         std::cout<<"gate 4"<<std::endl;
        for(int i=0; i<m_som.size(); ++i){
        for(int n=0; n<m_som[i].size(); ++n){
            double distToNodeSq = (m_winningNode->X()-m_som[i][n].X())*
                                (m_winningNode->X()-m_som[i][n].X())+
                                (m_winningNode->Y()-m_som[i][n].Y())*
                                (m_winningNode->Y()-m_som[i][n].Y());

            double widthSq = m_neighborhoodRadius * m_neighborhoodRadius;
            if(distToNodeSq < (widthSq)){
                m_influence = exp(-(distToNodeSq)/(2*widthSq));
                m_som[i][n].adjustWeights(data[curVector], m_lambda, m_influence);
                //m_som[i][n].adjustWeightsCuda(thrust::host_vector<double>(data[curVector]), m_lambda, m_influence,data[curVector].size());
            }
        }
        }
        m_lambda = constStartLearningRate * exp(-(double)m_iterationCount/m_numIterations);
        ++m_iterationCount;
    }
    else{
        m_done = true;
    } 
    return true;
}
__host__ bool Som::epoch(const std::vector<std::vector<double> > &data){
    if(data[0].size() != constSizeOfInputVector) return false;
    if(m_done) return true;
    if(--m_numIterations > 0){
        int curVector = RandInt(0, data.size()-1);
        m_winningNode = findBestMatch(data[curVector]);
        m_neighborhoodRadius = m_mapRadius * exp(-(double)m_iterationCount/m_timeConstant);

        for(int i=0; i<m_som.size(); ++i){
            for(int n=0; n<m_som[i].size(); ++n){
            double distToNodeSq = (m_winningNode->X()-m_som[i][n].X())*
                                (m_winningNode->X()-m_som[i][n].X())+
                                (m_winningNode->Y()-m_som[i][n].Y())*
                                (m_winningNode->Y()-m_som[i][n].Y());

            double widthSq = m_neighborhoodRadius * m_neighborhoodRadius;
            if(distToNodeSq < (widthSq)){
                m_influence = exp(-(distToNodeSq)/(2*widthSq));
                
                m_som[i][n].adjustWeights(data[curVector], m_lambda, m_influence);
                //m_som[i][n].adjustWeightsCuda(thrust::host_vector<double>(data[curVector]), m_lambda, m_influence,data[curVector].size());
            }
        }
        }
        m_lambda = constStartLearningRate * exp(-(double)m_iterationCount/m_numIterations);
        ++m_iterationCount;
    }
    else{
        m_done = true;
    } 
    return true;
}

__host__ Node* Som::findBestMatch(const std::vector<double> &vec){
    Node* winner = NULL;
    double lowestDistance = 999999;
    for(int i=0; i<m_som.size(); ++i){
        for(int n=0; n<m_som[i].size(); ++n){
            double dist = m_som[i][n].calcDistance(vec);
            if(dist < lowestDistance){
                lowestDistance = dist;
                winner = &m_som[i][n];
            }
        }
    }
    return winner;
}


void Som::render(){
//    print();
    
    for (int j=0; j<m_som.size(); ++j){
        for (int i=0; i<m_som[j].size(); ++i){
            glColor3f(m_som[j][i].getR(), m_som[j][i].getG(), m_som[j][i].getB());
            glBegin(GL_QUADS);
            glVertex3f(j, i, 0);            // upper left
            glVertex3f(j, i-1, 0);            // lower left
            glVertex3f(j+1, i-1, 0);            // upper right
            glVertex3f(j+1, i, 0);            // lower right
            glEnd();
        }
    }
}    

void Som::print(){
    std::cout<<std::endl;
    std::cout<<std::endl;
    std::cout<<"==================================================="<<std::endl;
    std::cout<<std::endl;
    for (int j=0; j<m_som.size(); ++j){
        std::vector<Node> row = m_som[j];
        for (int i=0; i<row.size(); ++i){
            std::cout<<"[ "<<row[i].getR()<<", "<<row[i].getG()<<", "<<row[i].getB()<<" ]    ";
        }
        std::cout<<std::endl;
    }
}

void Som::flipDone(){
    m_done = !m_done;
    if(m_done)
        std::cout<<"Stopped Training"<<std::endl;
    else
        std::cout<<"Training..."<<std::endl;
}
