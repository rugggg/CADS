#include "hip/hip_runtime.h"
#include "som.cuh"
#include "node.cuh"
#include "constants.h"
#include "utils.h"
#include <GL/glut.h>

void Som::create(int cellsUp,
                 int cellsAcross,
                 int numIterations)
{
    m_numIterations = numIterations;

    for(int row=0; row<cellsUp; ++row){
        std::vector<Node> m_row;
        for(int col=0; col<cellsAcross; ++col){
            m_row.push_back(Node(row,col,constSizeOfInputVector));
        }
        m_som.push_back(m_row);
    }
    m_mapRadius = max(constWindowWidth, constWindowHeight);
    m_timeConstant = m_numIterations/log(m_mapRadius);
    std::cout<<"tc: "<<m_timeConstant<<"  mr: "<<m_mapRadius<<std::endl;
}

bool Som::epoch(const std::vector<std::vector<double> > &data){
    //std::cout<<"=============================================="<<std::endl;
    //std::cout<<"==================== Epoch "<<m_iterationCount<<"==================="<<std::endl;

    if(data[0].size() != constSizeOfInputVector) return false;
    if(m_done) return true;
    if(--m_numIterations > 0){
        int curVector = RandInt(0, data.size()-1);
        //std::cout<<"Current Vector:: "<<std::endl;
        //std::cout<<data[curVector][0]<<" "<<data[curVector][1]<<" "<<data[curVector][2]<<std::endl;
        m_winningNode = findBestMatch(data[curVector]);
        m_neighborhoodRadius = m_mapRadius * exp(-(double)m_iterationCount/m_timeConstant);

        for(int i=0; i<m_som.size(); ++i){
            for(int n=0; n<m_som[i].size(); ++n){
            double distToNode = (m_winningNode->X()-m_som[i][n].X())*
                                (m_winningNode->X()-m_som[i][n].X())+
                                (m_winningNode->Y()-m_som[i][n].Y())*
                                (m_winningNode->Y()-m_som[i][n].Y());

            double widthSq = m_neighborhoodRadius * m_neighborhoodRadius;
            if(distToNode < (m_neighborhoodRadius * m_neighborhoodRadius)){
                m_influence = exp(-distToNode)/(2*widthSq)*10;
                m_som[i][n].adjustWeights(data[curVector], m_lambda, m_influence);
            }
        }
        }
        m_lambda = constStartLearningRate * exp(-(double)m_iterationCount/m_numIterations);
        ++m_iterationCount;
    }
    else{
        m_done = true;
    } 
    return true;
}

Node* Som::findBestMatch(const std::vector<double> &vec){
    Node* winner = NULL;
    double lowestDistance = 999999;
    for(int i=0; i<m_som.size(); ++i){
        for(int n=0; n<m_som[i].size(); ++n){
            double dist = m_som[i][n].calcDistance(vec);
            if(dist < lowestDistance){
                lowestDistance = dist;
                winner = &m_som[i][n];
            }
        }
    }
    return winner;
}

void Som::render(){
//    print();
    
    for (int j=0; j<m_som.size(); ++j){
        for (int i=0; i<m_som[j].size(); ++i){
            glColor3f(m_som[j][i].getR(), m_som[j][i].getG(), m_som[j][i].getB());
            glBegin(GL_QUADS);
            glVertex3f(j, i, 0);            // upper left
            glVertex3f(j, i-1, 0);            // lower left
            glVertex3f(j+1, i-1, 0);            // upper right
            glVertex3f(j+1, i, 0);            // lower right
            glEnd();
        }
    }
}    

void Som::print(){
    std::cout<<std::endl;
    std::cout<<std::endl;
    std::cout<<"==================================================="<<std::endl;
    std::cout<<std::endl;
    for (int j=0; j<m_som.size(); ++j){
        std::vector<Node> row = m_som[j];
        for (int i=0; i<row.size(); ++i){
            std::cout<<"[ "<<row[i].getR()<<", "<<row[i].getG()<<", "<<row[i].getB()<<" ]    ";
        }
        std::cout<<std::endl;
    }
}

void Som::flipDone(){
    m_done = !m_done;
    if(m_done)
        std::cout<<"Stopped Training"<<std::endl;
    else
        std::cout<<"Training..."<<std::endl;
}
