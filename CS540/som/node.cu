#include "hip/hip_runtime.h"
//node.cu

#include "node.cuh"
#include <stdlib.h>
#include <vector>
#include <time.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include "utils.h"

__global__ void cudaAdjustWeights(double *weight, double *target, const double lambda, const double influence){
    int i = (threadIdx.x + blockIdx.x * blockDim.x)-1;
    weight[i] += lambda* influence * (target[i] - weight[i]);
}


Node::Node(int x, int y, int numWeights){   
    for (int i = 0; i < numWeights; ++i){
        double r = RandFloat();
        m_weights.push_back(r);
        m_weightsCuda.push_back(r);
    }
    m_x = x;
    m_y = y;
}

double Node::calcDistance(const std::vector<double> &compareVector){
    double dis = 0;
    for(int i =0; i < m_weights.size(); ++i){
        dis += (compareVector[i] - m_weights[i]) * (compareVector[i] - m_weights[i]);
    }
    return sqrt(dis);
}

__host__ void Node::adjustWeights(const std::vector<double> &target, 
                         const double lambda, 
                         const double influence){
    for(int i=0; i < target.size(); ++i){
       m_weights[i] += lambda * influence * (target[i] - m_weights[i]);
    }
}

__host__ void Node::adjustWeightsCuda(thrust::host_vector<double> target, 
                         const double lambda, 
                         const double influence,
                         const int targetSize){
  
    thrust::device_vector<double> target_d = target;
    thrust::device_vector<double> weights_d = m_weightsCuda; 
    double *raw_weights = thrust::raw_pointer_cast(&weights_d[0]);
    double *raw_target = thrust::raw_pointer_cast(&target_d[0]);
    cudaAdjustWeights<<<3,1>>>(raw_weights, raw_target, lambda, influence);
    thrust::copy(weights_d.begin(), weights_d.end(), m_weightsCuda.begin());
    thrust::copy(weights_d.begin(), weights_d.end(), m_weights.begin());
}

